#include "hip/hip_runtime.h"
/*
This software is part of GPU Ocean. 

Copyright (C) 2018, 2019 SINTEF Digital
Copyright (C) 2018, 2019 Norwegian Meteorological Institute

This CUDA kernel implements the CDKLM numerical scheme
for the shallow water equations, described in
A. Chertock, M. Dudzinski, A. Kurganov & M. Lukacova-Medvidova
Well-Balanced Schemes for the Shallow Water Equations with Coriolis Forces,
Numerische Mathematik 2016

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "common.cu"




__device__ float3 CDKLM16_F_func(const float3 Q, const float g) {
    float3 F;

    F.x = Q.x*Q.y;                        //h*u
    F.y = Q.x*Q.y*Q.y + 0.5f*g*Q.x*Q.x;   //h*u*u + 0.5f*g*h*h;
    F.z = Q.x*Q.y*Q.z;                    //h*u*v;

    return F;
}







/**
  * Note that the input vectors are (h, u, v), thus not the regular
  * (h, hu, hv)
  */
__device__ float3 CDKLM16_flux(const float3 Qm, float3 Qp, const float g) {
    const float3 Fp = CDKLM16_F_func(Qp, g);
    const float up = Qp.y;         // u
    const float cp = sqrt(g*Qp.x); // sqrt(g*h)

    const float3 Fm = CDKLM16_F_func(Qm, g);
    const float um = Qm.y;         // u
    const float cm = sqrt(g*Qm.x); // sqrt(g*h)

    const float am = min(min(um-cm, up-cp), 0.0f); // largest negative wave speed
    const float ap = max(max(um+cm, up+cp), 0.0f); // largest positive wave speed

    float3 F;

    F.x = ((ap*Fm.x - am*Fp.x) + ap*am*(Qp.x-Qm.x))/(ap-am);
    F.y = ((ap*Fm.y - am*Fp.y) + ap*am*(Fp.x-Fm.x))/(ap-am);
    F.z = (Qm.y + Qp.y > 0) ? Fm.z : Fp.z; //Upwinding to be consistent

    return F;
}








__device__
float3 computeFFaceFlux(const int i, const int j, const int bx, const int nx_,
                float R[3][block_height+4][block_width+4],
                float Qx[3][block_height+2][block_width+2],
                float Hi[block_height+1][block_width+1],
                const float g_, const float coriolis_f, const float dx_,
                const int& bc_east_, const int& bc_west_) {
    const int l = j + 2; //Skip ghost cells (be consistent with reconstruction offsets)
    const int k = i + 1;

    // (u, v) reconstructed at a cell interface from the right (p) and left (m)
    // Variables to reconstruct h from u, v, K, L
    const float eta_bar_p = R[0][l][k+1];
    const float eta_bar_m = R[0][l][k  ];
    const float up = R[1][l][k+1];
    const float um = R[1][l][k  ];
    float vp = R[2][l][k+1];
    float vm = R[2][l][k  ];

    const float2 Rp = make_float2(up - 0.5f*Qx[0][j][i+1], vp - 0.5f*Qx[1][j][i+1]);
    const float2 Rm = make_float2(um + 0.5f*Qx[0][j][i  ], vm + 0.5f*Qx[1][j][i  ]);

    // H is RHx on the given face!
    const float H_face = 0.5f*( Hi[j][i] + Hi[j+1][i] );

    // Qx[2] is really dx*Kx
    const float Kx_p = Qx[2][j][i+1];
    const float Kx_m = Qx[2][j][i  ];
    
    // Fix west boundary for reconstruction of eta (corresponding to Kx)
    if ((bc_west_ == 1) && (bx + i + 2 == 2    )) { vm = -vm; }
    // Fix east boundary for reconstruction of eta (corresponding to Kx)
    if ((bc_east_ == 1) && (bx + i + 2 == nx_+2)) { vp = -vp; }
    
    // Reconstruct h
    //FIXME: CORIOLIS
    const float hp = eta_bar_p + H_face - (Kx_p + dx_*coriolis_f*vp)/(2.0f*g_);
    const float hm = eta_bar_m + H_face + (Kx_m + dx_*coriolis_f*vm)/(2.0f*g_);

    // Our flux variables Q=(h, u, v)
    const float3 Qp = make_float3(hp, Rp.x, Rp.y);
    const float3 Qm = make_float3(hm, Rm.x, Rm.y);

    // Computed flux
    return CDKLM16_flux(Qm, Qp, g_);
}




__device__
float3 computeGFaceFlux(const int i, const int j, const int by, const int ny_,
                float R[3][block_height+4][block_width+4],
                float Qy[3][block_height+2][block_width+2],
                float Hi[block_height+1][block_width+1],
                const float g_, const float coriolis_fm, const float coriolis_fp, const float dy_,
                const int& bc_north_, const int& bc_south_) {
    const int l = j + 1;
    const int k = i + 2; //Skip ghost cells
    // Q at interface from the right and left
    // Variables to reconstruct h from u, v, K, L
    const float eta_bar_p = R[0][l+1][k];
    const float eta_bar_m = R[0][l  ][k];
    float up = R[1][l+1][k];
    float um = R[1][l  ][k];
    const float vp = R[2][l+1][k];
    const float vm = R[2][l  ][k];

    const float2 Rp = make_float2(up - 0.5f*Qy[0][j+1][i], vp - 0.5f*Qy[1][j+1][i]);
    const float2 Rm = make_float2(um + 0.5f*Qy[0][j  ][i], vm + 0.5f*Qy[1][j  ][i]);

    // H is RHx on the given face!
    const float H_face = 0.5f*( Hi[j][i] + Hi[j][i+1] );

    // Qy[2] is really dy*Ly
    const float Ly_p = Qy[2][j+1][i];
    const float Ly_m = Qy[2][j  ][i];

    // Fix south boundary for reconstruction of eta (corresponding to Ly)
    if ((bc_south_ == 1) && (by + j + 2 == 2    )) { um = -um; }
    // Fix north boundary for reconstruction of eta (corresponding to Ly)
    if ((bc_north_ == 1) && (by + j + 2 == ny_+2)) { up = -up; }
    
    // Reconstruct h
    //FIXME: CORIOLIS
    const float hp = eta_bar_p + H_face - ( Ly_p - dy_*coriolis_fp*up)/(2.0f*g_);
    const float hm = eta_bar_m + H_face + ( Ly_m - dy_*coriolis_fm*um)/(2.0f*g_);

    // Our flux variables Q=(h, v, u)
    // Note that we swap u and v
    const float3 Qp = make_float3(hp, Rp.y, Rp.x);
    const float3 Qm = make_float3(hm, Rm.y, Rm.x);

    // Computed flux
    // Note that we swap back u and v
    const float3 flux = CDKLM16_flux(Qm, Qp, g_);
    return make_float3(flux.x, flux.z, flux.y);
}


__device__ 
void handleWallBC(
                const int& nx_, const int& ny_,
                const int& ti_, const int& tj_, 
                const int& tx_, const int& ty_, 
                const int& bc_north_, const int& bc_south_,
                const int& bc_east_, const int& bc_west_,
                float R[3][block_height+4][block_width+4]) {
    const int wall_bc = 1;

    const int i = tx_ + 2; //Skip local ghost cells, i.e., +2
    const int j = ty_ + 2;
        
    if (bc_north_ == wall_bc && tj_ == ny_+1) {
        R[0][j+1][i] =  R[0][j][i];
        R[1][j+1][i] =  R[1][j][i];
        R[2][j+1][i] = -R[2][j][i];

        R[0][j+2][i] =  R[0][j-1][i];
        R[1][j+2][i] =  R[1][j-1][i];
        R[2][j+2][i] = -R[2][j-1][i];
    }
    
    if (bc_south_ == wall_bc && tj_ == 2) {
        R[0][j-1][i] =  R[0][j][i];
        R[1][j-1][i] =  R[1][j][i];
        R[2][j-1][i] = -R[2][j][i];

        R[0][j-2][i] =  R[0][j+1][i];
        R[1][j-2][i] =  R[1][j+1][i];
        R[2][j-2][i] = -R[2][j+1][i];
    }
    
    if (bc_east_ == wall_bc && ti_ == nx_+1) {
        R[0][j][i+1] =  R[0][j][i];
        R[1][j][i+1] = -R[1][j][i];
        R[2][j][i+1] =  R[2][j][i];

        R[0][j][i+2] =  R[0][j][i-1];
        R[1][j][i+2] = -R[1][j][i-1];
        R[2][j][i+2] =  R[2][j][i-1];
    }
    
    if (bc_west_ == wall_bc && ti_ == 2) {
        R[0][j][i-1] =  R[0][j][i];
        R[1][j][i-1] = -R[1][j][i];
        R[2][j][i-1] =  R[2][j][i];

        R[0][j][i-2] =  R[0][j][i+1];
        R[1][j][i-2] = -R[1][j][i+1];
        R[2][j][i-2] =  R[2][j][i+1];
    }
}


texture<float, hipTextureType2D> angle_tex;

extern "C" {
__global__ void cdklm_swe_2D(
        const int nx_, const int ny_,
        const float dx_, const float dy_, const float dt_,
        const float g_,

        const float theta_,

        const float f_, //< Coriolis coefficient (f_ - beta_*y0)
        const float beta_, //< Coriolis force f_ + beta_*y

        const float r_, //< Bottom friction coefficient

        const int rk_order, // runge kutta order
        const int step_,    // runge kutta step

        //Input h^n
        float* eta0_ptr_, const int eta0_pitch_,
        float* hu0_ptr_, const int hu0_pitch_,
        float* hv0_ptr_, const int hv0_pitch_,

        //Output h^{n+1}
        float* eta1_ptr_, const int eta1_pitch_,
        float* hu1_ptr_, const int hu1_pitch_,
        float* hv1_ptr_, const int hv1_pitch_,

        //Bathymery
        float* Hi_ptr_, const int Hi_pitch_,
        float* Hm_ptr_, const int Hm_pitch_,

        //Wind stress parameters
        const float wind_stress_t_,

        // Boundary conditions (1: wall, 2: periodic, 3: open boundary (flow relaxation scheme))
        // Note: these are packed north, east, south, west boolean bits into an int
        const int boundary_conditions_) {


    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of block within domain
    const int bx = blockDim.x * blockIdx.x;
    const int by = blockDim.y * blockIdx.y;

    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 2;
    
    const float s = ti / (float) nx_;
    const float t = tj / (float) ny_;
    const float angle = tex2D(angle_tex, s, t);

    // Our physical variables
    // Input is [eta, hu, hv]
    // Will store [eta, u, v] (Note u and v are actually computed somewhat down in the code)
    __shared__ float R[3][block_height+4][block_width+4];

    // Our reconstruction variables
    //When computing flux along x-axis, we use
    //Qx = [u_x, v_x, K_x]
    //Then we reuse it as
    //Qx = [u_y, v_y, L_y]
    //to compute the y fluxes
    __shared__ float Qx[3][block_height+2][block_width+2];

    // Bathymetry
    __shared__ float  Hi[block_height+1][block_width+1];



    // theta_ = 1.5f;

    //Read into shared memory
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by + j, 0, ny_+3); // Out of bounds

        //Compute the pointer to current row in the arrays
        float* const eta_row = (float*) ((char*) eta0_ptr_ + eta0_pitch_*l);
        float* const hu_row = (float*) ((char*) hu0_ptr_ + hu0_pitch_*l);
        float* const hv_row = (float*) ((char*) hv0_ptr_ + hv0_pitch_*l);

        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx + i, 0, nx_+3); // Out of bounds

            R[0][j][i] = eta_row[k];
            R[1][j][i] = hu_row[k];
            R[2][j][i] = hv_row[k];
        }
    }
    __syncthreads();
    //Skip local ghost cells, i.e., +2
    const float hu = R[1][ty + 2][tx + 2];
    const float hv = R[2][ty + 2][tx + 2];


    // Read Hi into shared memory
    // Read intersections on all non-ghost cells
    for(int j=ty; j < block_height+1; j+=blockDim.y) {
        // Skip ghost cells and
        const int l = clamp(by+j+2, 2, ny_+2);
        float* const Hi_row = (float*) ((char*) Hi_ptr_ + Hi_pitch_*l);
        for(int i=tx; i < block_width+1; i+=blockDim.x) {
            const int k = clamp(bx+i+2, 2, nx_+2);

            Hi[j][i] = Hi_row[k];
        }
    }
    __syncthreads();
    const float Hm = 0.25f*(Hi[ty][tx]+Hi[ty+1][tx]+Hi[ty][tx+1]+Hi[ty+1][tx+1]);
    
    
    //Compute Coriolis terms needed for fluxes etc.
    // Global id should be including the 
    //FIXME CORIOLIS beta plane here
    const float coriolis_f_lower   = f_ + beta_ * (tj-0.5f)*dy_;
    const float coriolis_f_central = f_ + beta_ * (tj+0.5f)*dy_;
    const float coriolis_f_upper   = f_ + beta_ * (tj+1.5f)*dy_;



    //Fix boundary conditions
    //This must match code in CDKLM16.py:callKernel(...)
    const int bc_north = (boundary_conditions_ >> 24) & 0xFF;
    const int bc_south = (boundary_conditions_ >> 16) & 0xFF;
    const int bc_east = (boundary_conditions_ >> 8) & 0xFF;
    const int bc_west = (boundary_conditions_ >> 0) & 0xFF;
    
    if (boundary_conditions_ > 0) {
        // These boundary conditions are dealt with inside shared memory
        handleWallBC(nx_, ny_,
                ti, tj,
                tx, ty,
                bc_north, bc_south,
                bc_east, bc_west,
                R);
    }

    __syncthreads();



    //Create our "steady state" reconstruction variables (u, v)
    // K and L are never stored, but computed where needed.
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by+j, 0, ny_+3);
        float* const Hm_row = (float*) ((char*) Hm_ptr_ + Hm_pitch_*l);
        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx+i, 0, nx_+3);

            //const float h = R[0][j][i] + Hm[j][i]; // h = eta + H
            const float h = R[0][j][i] + Hm_row[k];
            R[1][j][i] /= h;
            R[2][j][i] /= h;
        }
    }
    __syncthreads();









    //Reconstruct slopes along x axis
    // Write result into shmem Qx = [u_x, v_x, K_x]
    // Qx is used as if its size was Qx[3][block_height][block_width + 2]
    for (int j=ty; j<block_height; j+=blockDim.y) {
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<block_width+2; i+=blockDim.x) {
            const int k = i + 1;

            const float left_eta   = R[0][l][k-1];
            const float center_eta = R[0][l][k  ];
            const float right_eta  = R[0][l][k+1];

            {
                const float left_u   = R[1][l][k-1];
                const float center_u = R[1][l][k  ];
                const float right_u  = R[1][l][k+1];
                Qx[0][j][i] = minmodSlope(left_u, center_u, right_u, theta_);
            }

            float left_v   = R[2][l][k-1];
            float center_v = R[2][l][k  ];
            float right_v  = R[2][l][k+1];
            Qx[1][j][i] = minmodSlope(left_v, center_v, right_v, theta_);
            
            // Enforce wall boundary conditions for Kx:
            int global_thread_id_x = bx + i + 1; // index including ghost cells'
            // Western BC
            if (bc_west == 1) {
                if (global_thread_id_x < 3    ) { left_v   = -left_v;   }
                if (global_thread_id_x < 2    ) { center_v = -center_v; }
            }
            // Eastern BC
            if (bc_east == 1) {
                if (global_thread_id_x > nx_  ) { right_v  = -right_v;  }
                if (global_thread_id_x > nx_+1) { center_v = -center_v; }
            }

            // by + j + 2 = global thread id + ghost cells
            //FIXME: CORIOLIS beta plane
            const float coriolis_f = f_ + beta_ * (by + l + 0.5f)*dy_;
            const float V_constant = dx_*coriolis_f/(2.0f*g_);

            // Qx[2] = Kx, which we need to find differently than ux and vx
            //FIXME: CORIOLIS
            const float backward = theta_*g_*(center_eta - left_eta   - V_constant*(center_v + left_v ) );
            const float central  =   0.5f*g_*(right_eta  - left_eta   - V_constant*(right_v + 2*center_v + left_v) );
            const float forward  = theta_*g_*(right_eta  - center_eta - V_constant*(center_v + right_v) );

            // Qx[2] is really dx*Kx
            Qx[2][j][i] = minmodRaw(backward, central, forward);

        }
    }
    __syncthreads();
    
    // Compute flux along x axis
    float3 flux_diff = (  computeFFaceFlux(tx+1, ty, bx, nx_, R, Qx, Hi,g_, coriolis_f_central, dx_, bc_north, bc_south) 
                        - computeFFaceFlux(tx  , ty, bx, nx_, R, Qx, Hi,g_, coriolis_f_central, dx_, bc_north, bc_south)) / dx_;
    __syncthreads();

    //Reconstruct slopes along y axis
    // Write result into shmem Qx = [u_y, v_y, L_y]
    // Qx is now used as if its size was Qx[3][block_height+2][block_width]

    for (int j=ty; j<block_height+2; j+=blockDim.y) {
        const int l = j + 1;
        for (int i=tx; i<block_width; i+=blockDim.x) {
            const int k = i + 2; //Skip ghost cells
            // Qy[2] = Ly, which we need to find differently than uy and vy
            const float lower_eta  = R[0][l-1][k];
            const float center_eta = R[0][l  ][k];
            const float upper_eta  = R[0][l+1][k];

            float lower_u  = R[1][l-1][k];
            float center_u = R[1][l  ][k];
            float upper_u  = R[1][l+1][k];
            Qx[0][j][i] = minmodSlope(lower_u, center_u, upper_u, theta_);

            {
                const float lower_v  = R[2][l-1][k];
                const float center_v = R[2][l  ][k];
                const float upper_v  = R[2][l+1][k];
                Qx[1][j][i] = minmodSlope(lower_v, center_v, upper_v, theta_);
            }

            // Enforce wall boundary conditions for Ly
            int global_thread_id_y = by + j + 1; // index including ghost cells
            // southern BC
            if (bc_south == 1) {
                if (global_thread_id_y < 3    ) { lower_u  = -lower_u;  }
                if (global_thread_id_y < 2    ) { center_u = -center_u; }
            }
            // northern BC
            if (bc_north == 1) {
                if (global_thread_id_y > ny_  ) { upper_u  = -upper_u;  }
                if (global_thread_id_y > ny_+1) { center_u = -center_u; }
            }
            
            //FIXME: CORIOLIS beta plane - change angle
            const float lower_coriolis_f  = f_ + beta_ * (by + l - 0.5f)*dy_;
            const float center_coriolis_f = f_ + beta_ * (by + l + 0.5f)*dy_;
            const float upper_coriolis_f  = f_ + beta_ * (by + l + 1.5f)*dy_;

            const float lower_fu  = lower_u*lower_coriolis_f;
            const float center_fu = center_u*center_coriolis_f;
            const float upper_fu  = upper_u*upper_coriolis_f;

            const float U_constant = dy_/(2.0f*g_);

            const float backward = theta_*g_*(center_eta - lower_eta  + U_constant*(center_fu + lower_fu ) );
            const float central  =   0.5f*g_*(upper_eta  - lower_eta  + U_constant*(upper_fu + 2*center_fu + lower_fu) );
            const float forward  = theta_*g_*(upper_eta  - center_eta + U_constant*(center_fu + upper_fu) );

            // Qy[2] is really dy*Ly
            Qx[2][j][i] = minmodRaw(backward, central, forward);
        }
    }
    __syncthreads();

    //Compute fluxes along the y axis    
    flux_diff = flux_diff + (  computeGFaceFlux(tx, ty+1, by, ny_, R, Qx, Hi, g_, coriolis_f_central,   coriolis_f_upper, dy_, bc_east, bc_west)
                             - computeGFaceFlux(tx, ty  , by, ny_, R, Qx, Hi, g_,   coriolis_f_lower, coriolis_f_central, dy_, bc_east, bc_west)) / dy_;
    __syncthreads();


    //Sum fluxes and advance in time for all internal cells
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;

        const float X = windStressX(wind_stress_t_, ti+0.5, tj+0.5, nx_, ny_);
        const float Y = windStressY(wind_stress_t_, ti+0.5, tj+0.5, nx_, ny_);

        // Bottom topography source terms!
        // -g*(eta + H)*(-1)*dH/dx   * dx
        const float RHxp = 0.5f*( Hi[ty][tx+1] + Hi[ty+1][tx+1] );
        const float RHxm = 0.5f*( Hi[ty][tx  ] + Hi[ty+1][tx  ] );
        const float st1 = g_*(R[0][j][i] + Hm)*(RHxp - RHxm);

        const float RHyp = 0.5f*( Hi[ty+1][tx] + Hi[ty+1][tx+1] );
        const float RHym = 0.5f*( Hi[ty  ][tx] + Hi[ty  ][tx+1] );
        const float st2 = g_*(R[0][j][i] + Hm)*(RHyp - RHym);

        //FIXME: CORIOLIS
        const float L1  = - flux_diff.x;
        const float L2  = - flux_diff.y + (X + coriolis_f_central*hv + st1/dx_);
        const float L3  = - flux_diff.z + (Y - coriolis_f_central*hu + st2/dy_);

        float* const eta_row = (float*) ((char*) eta1_ptr_ + eta1_pitch_*tj);
        float* const hu_row  = (float*) ((char*) hu1_ptr_  +  hu1_pitch_*tj);
        float* const hv_row  = (float*) ((char*) hv1_ptr_  +  hv1_pitch_*tj);

        if (rk_order < 3) {

            const float C = 2.0f*r_*dt_/(R[0][j][i] + Hm);

            if  (step_ == 0) {
                //First step of RK2 ODE integrator

                eta_row[ti] =  R[0][j][i] + dt_*L1;
                hu_row[ti]  = (hu + dt_*L2) / (1.0f + C);
                hv_row[ti]  = (hv + dt_*L3) / (1.0f + C);
            }
            else if (step_ == 1) {
                //Second step of RK2 ODE integrator

                //First read Q^n
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                //Compute Q^n+1
                const float eta_b = 0.5f*(eta_a + (R[0][j][i] + dt_*L1));
                const float hu_b  = 0.5f*( hu_a + (hu + dt_*L2));
                const float hv_b  = 0.5f*( hv_a + (hv + dt_*L3));


                //Write to main memory
                eta_row[ti] = eta_b;
                hu_row[ti]  =  hu_b / (1.0f + 0.5f*C);
                hv_row[ti]  =  hv_b / (1.0f + 0.5f*C);

            }
        }


        else if (rk_order == 3) {
            // Third order Runge Kutta - only valid if r_ = 0.0 (no friction)

            if (step_ == 0) {
                //First step of RK3 ODE integrator
                // q^(1) = q^n + dt*L(q^n)

                eta_row[ti] =  R[0][j][i] + dt_*L1;
                hu_row[ti]  = (hu + dt_*L2);
                hv_row[ti]  = (hv + dt_*L3);

            } else if (step_ == 1) {
                // Second step of RK3 ODE integrator
                // Q^(2) = 3/4 Q^n + 1/4 ( Q^(1) + dt*L(Q^(1)) )
                // Q^n is here in h1, but will be used in next iteration as well --> write to h0

                // First read Q^n:
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                // Compute Q^(2):
                const float eta_b = 0.75f*eta_a + 0.25f*(R[0][j][i] + dt_*L1);
                const float hu_b  = 0.75f* hu_a + 0.25f*(hu + dt_*L2);
                const float hv_b  = 0.75f* hv_a + 0.25f*(hv + dt_*L3);

                // Write output to the input buffer:
                float* const eta_out_row = (float*) ((char*) eta0_ptr_ + eta0_pitch_*tj);
                float* const hu_out_row  = (float*) ((char*)  hu0_ptr_ +  hu0_pitch_*tj);
                float* const hv_out_row  = (float*) ((char*)  hv0_ptr_ +  hv0_pitch_*tj);
                eta_out_row[ti] = eta_b;
                hu_out_row[ti]  =  hu_b;
                hv_out_row[ti]  =  hv_b;

            } else if (step_ == 2) {
                // Third step of RK3 ODE integrator
                // Q^n+1 = 1/3 Q^n + 2/3 (Q^(2) + dt*L(Q^(2))

                // First read Q^n:
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                // Compute Q^n+1:
                const float eta_b = (eta_a + 2.0f*(R[0][j][i] + dt_*L1)) / 3.0f;
                const float hu_b  = ( hu_a + 2.0f*(hu + dt_*L2)) / 3.0f;
                const float hv_b  = ( hv_a + 2.0f*(hv + dt_*L3)) / 3.0f;

                //Write to main memory
                eta_row[ti] = eta_b;
                hu_row[ti]  =  hu_b;
                hv_row[ti]  =  hv_b;
            }
        }
    }



}

}

